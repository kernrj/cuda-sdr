#include "hip/hip_runtime.h"
/*
 * Copyright 2022 Rick Kern <kernrj@gmail.com>
 *
 *  Licensed under the Apache License, Version 2.0 (the "License");
 *  you may not use this file except in compliance with the License.
 *  You may obtain a copy of the License at
 *
 *      http://www.apache.org/licenses/LICENSE-2.0
 *
 *  Unless required by applicable law or agreed to in writing, software
 *  distributed under the License is distributed on an "AS IS" BASIS,
 *  WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 *  See the License for the specific language governing permissions and
 *  limitations under the License.
 */

#include <hip/hip_complex.h>
#include <hip/hip_runtime.h>

#include "CudaBuffers.h"
#include "CudaDevicePushPop.h"
#include "Multiply.h"
#include "cuComplexOperatorOverloads.cuh"
#include "cuda_util.h"

using namespace std;

const size_t MultiplyCcc::mAlignment = 32;

template <class IN1_T, class IN2_T, class OUT_T>
__global__ void k_Multiply(const IN1_T* in1, const IN2_T* in2, OUT_T* out) {
  const size_t x = blockDim.x * blockIdx.x + threadIdx.x;

  out[x] = in1[x] * in2[x];
}

MultiplyCcc::MultiplyCcc(int32_t cudaDevice, hipStream_t cudaStream)
    : mCudaDevice(cudaDevice), mCudaStream(cudaStream) {}

Buffer MultiplyCcc::requestBuffer(size_t port, size_t numBytes) {
  if (port >= mInputBuffers.size()) {
    throw runtime_error("Port [" + to_string(port) + "] is out of range");
  }

  CudaDevicePushPop setAndRestore(mCudaDevice);
  ensureMinCapacityAligned(
      &mInputBuffers[port],
      numBytes,
      mAlignment * sizeof(hipComplex),
      mCudaStream);

  return mInputBuffers[port].sliceRemainingUnowned();
}

void MultiplyCcc::commitBuffer(size_t port, size_t numBytes) {
  if (port >= 2) {
    throw runtime_error("Port [" + to_string(port) + "] is out of range");
  }

  OwnedBuffer& buffer = mInputBuffers[port];

  const size_t newEndIndex = buffer.end + numBytes;

  if (newEndIndex > buffer.capacity) {
    throw runtime_error(
        "Committed byte count [" + to_string(numBytes) + "] at offset ["
        + to_string(buffer.end) + "] exceeds capacity ["
        + to_string(buffer.capacity) + "]");
  }

  buffer.end += numBytes;
}

size_t MultiplyCcc::getOutputDataSize(size_t port) {
  return getAvailableNumInputElements() * sizeof(hipComplex);
}

size_t MultiplyCcc::getAvailableNumInputElements() const {
  const size_t port0NumElements = mInputBuffers[0].used() / sizeof(hipComplex);
  const size_t port1NumElements = mInputBuffers[1].used() / sizeof(hipComplex);
  const size_t numInputElements = min(port0NumElements, port1NumElements);

  return numInputElements;
}

size_t MultiplyCcc::getOutputSizeAlignment(size_t port) {
  return mAlignment * sizeof(hipComplex);
}

void MultiplyCcc::readOutput(Buffer* portOutputs, size_t portOutputCount) {
  if (portOutputCount < 1) {
    throw runtime_error("One output port is required");
  }

  CudaDevicePushPop setAndRestore(mCudaDevice);

  const size_t numInputElements = getAvailableNumInputElements();
  Buffer& outputBuffer = portOutputs[0];
  const size_t maxNumOutputElements =
      outputBuffer.remaining() / sizeof(hipComplex);

  const size_t maxUnalignedNumElementsToProcess =
      min(numInputElements, maxNumOutputElements);

  const size_t numBlocks = maxUnalignedNumElementsToProcess / mAlignment;
  const size_t processNumInputElements = numBlocks * mAlignment;

  const dim3 blocks = dim3(numBlocks);
  const dim3 threads = dim3(mAlignment);

  k_Multiply<hipComplex, hipComplex, hipComplex>
      <<<blocks, threads, 0, mCudaStream>>>(
          mInputBuffers[0].readPtr<hipComplex>(),
          mInputBuffers[1].readPtr<hipComplex>(),
          portOutputs[0].writePtr<hipComplex>());

  const size_t writtenNumBytes = processNumInputElements * sizeof(hipComplex);
  portOutputs[0].end += writtenNumBytes;
}
