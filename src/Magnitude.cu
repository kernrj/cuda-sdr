#include "hip/hip_runtime.h"
/*
 * Copyright 2022 Rick Kern <kernrj@gmail.com>
 *
 *  Licensed under the Apache License, Version 2.0 (the "License");
 *  you may not use this file except in compliance with the License.
 *  You may obtain a copy of the License at
 *
 *      http://www.apache.org/licenses/LICENSE-2.0
 *
 *  Unless required by applicable law or agreed to in writing, software
 *  distributed under the License is distributed on an "AS IS" BASIS,
 *  WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 *  See the License for the specific language governing permissions and
 *  limitations under the License.
 */

#include <hip/hip_complex.h>
#include <hip/hip_runtime.h>

#include "CudaBuffers.h"
#include "CudaDevicePushPop.h"
#include "Magnitude.h"
#include "cuComplexOperatorOverloads.cuh"
#include "cuda_util.h"

using namespace std;

const size_t Magnitude::mAlignment = 32;

__global__ void k_Magnitude(const hipComplex* in, float* out) {
  const size_t x = blockDim.x * blockIdx.x + threadIdx.x;

  hipComplex input = in[x];
  out[x] = hypotf(input.x, input.y);
}

Magnitude::Magnitude(int32_t cudaDevice, hipStream_t cudaStream)
    : mCudaDevice(cudaDevice), mCudaStream(cudaStream) {}

Buffer Magnitude::requestBuffer(size_t port, size_t numBytes) {
  if (port >= 1) {
    throw runtime_error("Port [" + to_string(port) + "] is out of range");
  }

  CudaDevicePushPop setAndRestore(mCudaDevice);
  ensureMinCapacityAligned(
      &mInputBuffer,
      numBytes,
      mAlignment * sizeof(hipComplex),
      mCudaStream);

  return mInputBuffer.sliceRemainingUnowned();
}

void Magnitude::commitBuffer(size_t port, size_t numBytes) {
  if (port >= 1) {
    throw runtime_error("Port [" + to_string(port) + "] is out of range");
  }

  const size_t newEndIndex = mInputBuffer.end + numBytes;

  if (newEndIndex > mInputBuffer.capacity) {
    throw runtime_error(
        "Committed byte count [" + to_string(numBytes) + "] at offset ["
        + to_string(mInputBuffer.end) + "] exceeds capacity ["
        + to_string(mInputBuffer.capacity) + "]");
  }

  mInputBuffer.end += numBytes;
}

size_t Magnitude::getOutputDataSize(size_t port) {
  return getAvailableNumInputElements() * sizeof(float);
}

size_t Magnitude::getAvailableNumInputElements() const {
  return mInputBuffer.used() / sizeof(hipComplex);
}

size_t Magnitude::getOutputSizeAlignment(size_t port) {
  return mAlignment * sizeof(float);
}

void Magnitude::readOutput(Buffer* portOutputs, size_t portOutputCount) {
  if (portOutputCount < 1) {
    throw runtime_error("One output port is required");
  }

  CudaDevicePushPop setAndRestore(mCudaDevice);

  const size_t numInputElements = getAvailableNumInputElements();
  Buffer& outputBuffer = portOutputs[0];
  const size_t maxNumOutputElements = outputBuffer.remaining() / sizeof(float);

  const size_t maxUnalignedNumElementsToProcess =
      min(numInputElements, maxNumOutputElements);

  const size_t numBlocks = maxUnalignedNumElementsToProcess / mAlignment;
  const size_t processNumInputElements = numBlocks * mAlignment;

  const dim3 blocks = dim3(numBlocks);
  const dim3 threads = dim3(mAlignment);

  k_Magnitude<<<blocks, threads, 0, mCudaStream>>>(
      mInputBuffer.readPtr<hipComplex>(),
      portOutputs[0].writePtr<float>());

  const size_t writtenNumBytes = processNumInputElements * sizeof(float);
  portOutputs[0].end += writtenNumBytes;
}
