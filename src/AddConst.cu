#include "hip/hip_runtime.h"
/*
 * Copyright 2022 Rick Kern <kernrj@gmail.com>
 *
 *  Licensed under the Apache License, Version 2.0 (the "License");
 *  you may not use this file except in compliance with the License.
 *  You may obtain a copy of the License at
 *
 *      http://www.apache.org/licenses/LICENSE-2.0
 *
 *  Unless required by applicable law or agreed to in writing, software
 *  distributed under the License is distributed on an "AS IS" BASIS,
 *  WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 *  See the License for the specific language governing permissions and
 *  limitations under the License.
 */

#include <hip/hip_runtime.h>

#include "AddConst.h"
#include "CudaBuffers.h"
#include "CudaDevicePushPop.h"
#include "cuda_util.h"

using namespace std;

const size_t AddConst::mAlignment = 32;

__global__ void k_AddConst(const float* in, float addConst, float* out) {
  const size_t x = blockDim.x * blockIdx.x + threadIdx.x;

  out[x] = addConst + in[x];
}

AddConst::AddConst(float addConst, int32_t cudaDevice, hipStream_t cudaStream)
    : mAddConst(addConst), mCudaDevice(cudaDevice), mCudaStream(cudaStream) {}

Buffer AddConst::requestBuffer(size_t port, size_t numBytes) {
  if (port >= 1) {
    throw runtime_error("Port [" + to_string(port) + "] is out of range");
  }

  CudaDevicePushPop setAndRestore(mCudaDevice);
  ensureMinCapacityAligned(
      &mInputBuffer,
      numBytes,
      mAlignment * sizeof(float),
      mCudaStream);

  return mInputBuffer.sliceRemainingUnowned();
}

void AddConst::commitBuffer(size_t port, size_t numBytes) {
  if (port >= 1) {
    throw runtime_error("Port [" + to_string(port) + "] is out of range");
  }

  OwnedBuffer& buffer = mInputBuffer;

  const size_t newEndIndex = buffer.end + numBytes;

  if (newEndIndex > buffer.capacity) {
    throw runtime_error(
        "Committed byte count [" + to_string(numBytes) + "] at offset ["
        + to_string(buffer.end) + "] exceeds capacity ["
        + to_string(buffer.capacity) + "]");
  }

  buffer.end += numBytes;
}

size_t AddConst::getOutputDataSize(size_t port) {
  return getAvailableNumInputElements() * sizeof(float);
}

size_t AddConst::getAvailableNumInputElements() const {
  return mInputBuffer.used() / sizeof(float);
}

size_t AddConst::getOutputSizeAlignment(size_t port) {
  return mAlignment * sizeof(float);
}

void AddConst::readOutput(Buffer* portOutputs, size_t portOutputCount) {
  if (portOutputCount < 1) {
    throw runtime_error("One output port is required");
  }

  CudaDevicePushPop setAndRestore(mCudaDevice);

  const size_t numInputElements = getAvailableNumInputElements();
  Buffer& outputBuffer = portOutputs[0];
  const size_t maxNumOutputElements = outputBuffer.remaining() / sizeof(float);

  const size_t maxUnalignedNumElementsToProcess =
      min(numInputElements, maxNumOutputElements);

  const size_t numBlocks = maxUnalignedNumElementsToProcess / mAlignment;
  const size_t processNumInputElements = numBlocks * mAlignment;

  const dim3 blocks = dim3(numBlocks);
  const dim3 threads = dim3(mAlignment);

  k_AddConst<<<blocks, threads, 0, mCudaStream>>>(
      mInputBuffer.readPtr<float>(),
      mAddConst,
      portOutputs[0].writePtr<float>());

  const size_t writtenNumBytes = processNumInputElements * sizeof(float);
  portOutputs[0].end += writtenNumBytes;
}
